#include <iostream>
#include <hip/hip_runtime.h>

#include <math.h>

#define BS 16
#define N 50
#define R 20
#define K 3

// N - dlugosc tablicy
// R - dlugosc promienia zliczania
// BS - wielkosc bloku

__global__ void calculateGlobal(int* input_tab, int* output_tab, int Nx, int Rx, int Kx) {
    int row = (blockIdx.y * blockDim.y + threadIdx.y) * Kx - 1;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int output_tab_size = Nx - 2 * Rx;

    for (int k_iter = 0; k_iter < Kx; k_iter++) {
        row++;

        if (col < output_tab_size && row < output_tab_size) {
            int sum = 0;

            // Zliczanie sumy elementów w zasięgu promienia R
            for (int i = -Rx; i <= Rx; i++) {
                for (int j = -Rx; j <= Rx; j++) {
                    sum += input_tab[(row + j + Rx) * Nx + col + Rx + i];
                }
            }

            // Zapisywanie wyników sum do tablicy wynikowej
            output_tab[row * output_tab_size + col] = sum;
        }
    }
}

__global__ void calculateShared(int* input_tab, int* output_tab, int Nx, int Rx, int Kx) {
    int row = (blockIdx.y * blockDim.y + threadIdx.y);
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int output_tab_size = Nx - 2 * Rx;
    int calculation_radius_range = 2 * Rx + 1;
    const int shared_input_tab_size = BS + 2 * R + 1;

    for (int k_iter = 0; k_iter < Kx; k_iter++) {
        int row_offset = row;
        int col_offset = col + k_iter * output_tab_size;

        if (col_offset < output_tab_size && row_offset < output_tab_size) {
            // Inicjalizacja tablicy pamięci współdzielonej
            __shared__ int shared_input_tab[shared_input_tab_size * shared_input_tab_size];

            // Wczytanie danych do pamięci współdzielonej przez wątek (0, 0)
            if (threadIdx.x == 0 && threadIdx.y == 0) {
                for (int i = 0; i < shared_input_tab_size; i++) {
                    for (int j = 0; j < shared_input_tab_size; j++) {
                        shared_input_tab[i * shared_input_tab_size + j] = input_tab[(row_offset + i) * Nx + col_offset + j];
                    }
                }
            }

            // Synchronizacja wątków po wczytaniu danych do pamięci współdzielonej
            __syncthreads();

            // Zliczanie sumy elementów w zasięgu promienia R
            int sum = 0;

            for (int i = 0; i < calculation_radius_range; i++) {
                for (int j = 0; j < calculation_radius_range; j++) {
                    sum += shared_input_tab[(threadIdx.y + i) * shared_input_tab_size + threadIdx.x + j];
                }
            }

            // Zapisywanie wyników sum do tablicy wynikowej
            output_tab[row_offset * output_tab_size + col_offset] = sum;
        }
    }
}

// Wypelnianie tablicy liczbami 0 - 100
void fill_table(int* table, int tab_size) {
    for (int i = 0; i < tab_size; i++) {
        table[i] = i % 100;
    }
}



void print_table(int* table, int tab_size) {
    for (int i = 0; i < tab_size; i++) {

        if (i % int(sqrt(tab_size)) == 0) {
            std::cout << "\n";
        }
        std::cout << table[i] << " ";
    }
}




int main() {
    const int input_tab_size = N * N;
    const int output_tab_size = (N - 2 * R) * (N - 2 * R);

    // Alokuje pamięć na GPU
    int* device_input;
    int* device_output;

    hipHostAlloc((void**)&device_input, input_tab_size * sizeof(int), hipHostMallocMapped);
    hipHostAlloc((void**)&device_output, output_tab_size * sizeof(int), hipHostMallocMapped);

    fill_table(device_input, input_tab_size);

    // Konfiguracja wątków i bloków
    dim3 blockSize(BS, BS);
    dim3 gridSize((N - 2 * R + blockSize.x - 1) / blockSize.x, (N - 2 * R + blockSize.y - 1) / blockSize.y);

    // Wywołanie kernela na GPU
    calculateGlobal <<<gridSize, blockSize>>> (device_input, device_output, N, R, K);

    hipDeviceSynchronize();

    //print_table(device_output, output_tab_size);

     // Zwolnienie pamięci na GPU
    hipHostFree(device_input);
    hipHostFree(device_output);

    return 0;
}
